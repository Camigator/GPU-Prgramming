// Cameron O'Neal


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 10240;
const int blocksize = 1024;

// initalize Array's 

	int a[N];
 	int b[N];
	int c[N];



__global__
void Addition(int *a, int * b, int *c)
{
	//threads 0 - 1023 are doing computation in 4 different blocks
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];		
	
		
} 
int main()
{	
		
	
	int *ad;		//pointer to a array
 	int *bd;		//pointer to b array
	int *cd;		//pointer to c array
	
	// intializing array a & b to correct values
	for(int i = 0; i < N; i++)
	{
		a[i] = 2 * i;
		b[i] = 2*i + 1;
	}
	
	
 	const int isize = N*sizeof(int);
	
	//allocate pointers to global mem with size = Isize
 	hipMalloc( (void**)&ad, isize );
 	hipMalloc( (void**)&bd, isize );
 	hipMalloc( (void**)&cd, isize );

	// memory Data Transfer
 	hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
 	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );
	hipMemcpy( cd, b, isize, hipMemcpyHostToDevice );

	
 	dim3 dimGrid( 2, 1 ); 		
	dim3 dimBlock( blocksize, 1 );

	// calls addition function -> sending pointer to arrays
	for(int i = 0; i < 5 ; i++)
	{	
		
		Addition<<<dimGrid, dimBlock>>>(ad, bd, cd);
	}
	
	
	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );
	printf("%d\n%d\n", c[0], c[N-1]);

	dim3 dimGrid2( 10, 1 ); 	
	Addition<<<dimGrid2, dimBlock>>>(ad, bd, cd);
	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );
	
	

	// frees poiters
 	hipFree( ad );
	hipFree( bd );
	hipFree( cd );
	
	
	

	// print out values in c[0] , c[N-1]
 	printf("%d\n%d", c[0], c[N-1]);
	
	
	 
 	return EXIT_SUCCESS;
}
