// Cameron O'Neal


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


const double PI = 3.14159265;

//const int Samples = 8192;
const int N = 8;
const int Threadcount = 4;

// initalize Array's 

	
__global__
void FFT(double * R, double * img, double * XR, double * XI, double *c , double * s)
{	
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	

	
	XR[2*i] = (R[2*i] * c[2*i] - img[2*i] * s[2*i]*-1);
	XI[2*i] = (c[2*i]*img[2*i] - R[2*i] * s[2*i]);

	XR[2*i+1] = (R[2*i+1] * c[2*i+1] - img[2*i+1] * s[2*i+1]*-1);	
	XI[2*i+1] = (c[2*i+1]*img[2*i+1] - R[2*i+1] * s[2*i+1]);
	
	
}



int main()
{	
		
	
	double * ad;		//pointer to REAL array
 	double * bd;		//pointer to IMAGINARY array
	double * cd;		//pointer to XR array
	double * dd; 		//pointer to IR array
	double * c;
	double * s;
	
	
	// intializing array REAL & IMAGINARY to correct values
	double REAL[N];
	double IMAGINARY[N];
	double Cosine[N];
	double Sine[N];
	double XR[N];
	double XI[N];
	
	REAL[0] = 3.6; IMAGINARY[0] = 2.6;
	REAL[1] = 2.9; IMAGINARY[1] = 6.3;
	REAL[2] = 5.6; IMAGINARY[2] = 4.0;
	REAL[3] = 4.8; IMAGINARY[3] = 9.1;
	REAL[4] = 3.3; IMAGINARY[4] = 0.4;
	REAL[5] = 5.9; IMAGINARY[5] = 4.8;
	REAL[6] = 5.0; IMAGINARY[6] = 2.6;
	REAL[7] = 4.3; IMAGINARY[7] = 4.1;
	
		for(int i = 8; i < N; i++)
    	{  
			REAL[i] = 0;
		   IMAGINARY[i] = 0;
    	}
		
	for(int i = 0; i < N; i++)
		{  
			XR[i] = 0;
		   XI[i] = 0;
		}	
		
	double b;
	
	for (int i = 0; i < N/2 ; i++)
	{
		b= (2*PI*2*i)/N;
		Cosine[2*i] = cos(b);
		Sine[2*i] = sin(b);
		
		b= (2*PI*((2*i)+1))/N;
		Cosine[2*i + 1] = cos(b);
		Sine[2*i + 1] = sin(b);

	}
	
	

	const double isize = N*sizeof(double);
	
	//allocate pointers to global mem with size = Isize
 	hipMalloc( (void**)&ad, isize );
 	hipMalloc( (void**)&bd, isize );
 	hipMalloc( (void**)&cd, isize );
	hipMalloc( (void**)&dd, isize );
	hipMalloc( (void**)&c, isize );
	hipMalloc( (void**)&s, isize );
	

	// memory Data Transfer
 	hipMemcpy( ad, REAL, isize, hipMemcpyHostToDevice );
 	hipMemcpy( bd, IMAGINARY, isize, hipMemcpyHostToDevice );
	hipMemcpy( cd, XR, isize, hipMemcpyHostToDevice );
	hipMemcpy( dd, XI, isize, hipMemcpyHostToDevice );
	hipMemcpy( c, Cosine, isize, hipMemcpyHostToDevice );
	hipMemcpy( s, Sine, isize, hipMemcpyHostToDevice );
	

	
 	dim3 dimGrid( 4, 1 ); 		
	dim3 dimBlock( Threadcount, 1 );
	
	
	FFT<<<dimGrid, dimBlock>>>(ad, bd, cd, dd, c, s);
	hipMemcpy( XR, cd, isize, hipMemcpyDeviceToHost );
	hipMemcpy( XI, dd, isize, hipMemcpyDeviceToHost );
	
	printf("================================== \n" );
	for( int i = 0; i < N ; i++)
	{
			
			printf("XR[%d] : %f         XI[%d] : %f \n", i, XR[i], i, XI[i]);
			printf("================================== \n" );
	}
	
	
	

	// frees poiters
 	hipFree( ad );
	hipFree( bd );
	hipFree( cd );
	hipFree( dd ) ;
	hipFree( c ) ;
	hipFree( s ) ;
	
	
	 
 	return EXIT_SUCCESS;
}






