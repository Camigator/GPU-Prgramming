// Cameron O'Neal


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 4096;
const int blocksize = 1024;

// initalize Array's 

	int a[N];
 	int b[N];
	int c[N];

__global__
void Addition(int *a, int * b, int *c)
{
	//threads 0 - 1023 are doing computation in 4 different blocks
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];
			
	
		
} 

int main()
{
	
	
		
	int Sum = 0;	//add all integer in c array
	int *ad;		//pointer to a array
 	int *bd;		//pointer to b array
	int *cd;		//pointer to c array
	
	// intializing array a & b to correct values
	for(int i = 0; i < 4096; i++)
	{
		a[i] = i;
		b[i] = 4095 + i;
	}
	
	
 	const int isize = N*sizeof(int);
	
	//allocate pointers to global mem with size = Isize
 	hipMalloc( (void**)&ad, isize );
 	hipMalloc( (void**)&bd, isize );
 	hipMalloc( (void**)&cd, isize );

	// memory Data Transfer
 	hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
 	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );
	hipMemcpy( cd, b, isize, hipMemcpyHostToDevice );

	
 	dim3 dimGrid( 4, 1 ); 	
	dim3 dimBlock( blocksize, 1 );

	// calls addition function -> sending pointer to arrays
 	Addition<<<dimGrid, dimBlock>>>(ad, bd, cd);

	//retreaving data from host
 	hipMemcpy( c, cd, isize ,hipMemcpyDeviceToHost );

	// frees poiters
 	hipFree( ad );
	hipFree( bd );
	hipFree( cd );
	
	// adding the sum in array c
	for(int i = 0; i < 4096; i++)
	{
		Sum += c[i];
	}	

	// print out values in c[0] , c[4095] , and sum of all values.
 	printf("%d\n%d\n%d", c[0], c[4095], Sum);
	 
 	return EXIT_SUCCESS;
}
