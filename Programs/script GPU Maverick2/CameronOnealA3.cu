// Cameron O'Neal


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 10240;
const int blocksize = 1024;

// initalize Array's 

	int a[N];
 	int b[N];
	int c[N];

// B2 in the kernel that snakes through the array as we take 
__global__
void B2(int *a, int * b, int *c, int x)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	i = i + x*2048;
	c[i] = a[i] * b[i];		
		
} 


__global__
void B3(int *a, int * b, int *c)
{
			
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		for(int k = 0; k<5; k++)
		c[i] = a[i] * b[i];
			
	
		
} 
__global__
void Addition(int *a, int * b, int *c)
{
	//threads 0 - 1023 are doing computation in  different blocks
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];		
	
		
} 
int main()
{	
		
	
	int *ad;		//pointer to a array
 	int *bd;		//pointer to b array
	int *cd;		//pointer to c array
	
	// intializing array a & b to correct values
	for(int i = 0; i < N; i++)
	{
		a[i] = 2 * i;
		b[i] = 2*i + 1;
	}
	
	
 	const int isize = N*sizeof(int);
	
	//allocate pointers to global mem with size = Isize
 	hipMalloc( (void**)&ad, isize );
 	hipMalloc( (void**)&bd, isize );
 	hipMalloc( (void**)&cd, isize );

	// memory Data Transfer
 	hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
 	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );
	hipMemcpy( cd, b, isize, hipMemcpyHostToDevice );

	
 	dim3 dimGrid( 2, 1 ); 		
	dim3 dimBlock( blocksize, 1 );

	for(int k = 0; k < 5 ; k++)
	B2<<<dimGrid, dimBlock>>>(ad, bd, cd, k);
	
	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );	
	printf("2 blocks  Non Cyclic ( C[0], C[10239] ) = ( %d , %d )\n", c[0], c[N-1]);
	
	
	
	B3<<<dimGrid, dimBlock>>>(ad, bd, cd);	
	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );
	printf("2 blocks Cyclic ( C[0], C[10239] ) = ( %d , %d) \n", c[0], c[N-1]);



	dim3 dimGrid2( 10, 1 ); 	
	Addition<<<dimGrid2, dimBlock>>>(ad, bd, cd);
	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );
	// print out values in c[0] , c[N-1]
 	printf("10 blocks ( C[0], C[10239] ) = ( %d , %d)\n", c[0], c[N-1]);
	
	

	// frees poiters
 	hipFree( ad );
	hipFree( bd );
	hipFree( cd );
	
	
	

	
	
	 
 	return EXIT_SUCCESS;
}
