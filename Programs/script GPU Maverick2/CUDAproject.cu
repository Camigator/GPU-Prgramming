#include "hip/hip_runtime.h"
//sbatch Project_Script_Par
#include <stdio.h>
#include "timer.h"

const int N = 1000;
const int blocksize = 38;

__global__ void wordSearchSimple(char *Data, int dataLength, char *Target, int targetLen, int *count)
{
	int startIndex = threadIdx.x;
	int fMatch = 1;
    
    for (int i=0; i < targetLen; i++)
    {	
        if (Data[startIndex] != Target[i])
            fMatch = 0;
		
		printf( " Thread : %d , fMatch %d , Target : %c , Data [%d] : %c \n " , threadIdx.x, fMatch , Target[i] , threadIdx.x, Data[threadIdx.x]);
    }
	
    if (fMatch == 1)
        count[0]++;
}
int main()
{
    char *CharacterArray;
    char *SuckMyAss;
    int *IntArray;		
    int count[1] = {0};
    FILE *infile;
    char *BookData;
	char WORD[3] = {'h','e','y'};
    int numbytes;
	double start, finish, elapsed;
	
    const int booksize = N*sizeof(char);
    const int intsize = sizeof(int);
    infile = fopen("input.txt", "r");
    fseek(infile, 0L, SEEK_END);
    numbytes = ftell(infile);
    fseek(infile, 0L, SEEK_SET);
    BookData = (char*)calloc(numbytes, sizeof(char));
    fread(BookData, sizeof(char), numbytes, infile);
    fclose(infile);
    hipMalloc( (void**)&CharacterArray, booksize );
    hipMalloc( (void**)&SuckMyAss, booksize );
    hipMalloc( (void**)&IntArray, intsize );
    hipMemcpy( CharacterArray, BookData, booksize, hipMemcpyHostToDevice ); 
    hipMemcpy( SuckMyAss, WORD, booksize, hipMemcpyHostToDevice ); 		
    hipMemcpy( IntArray, count, intsize, hipMemcpyHostToDevice );
    dim3 dimGrid(1, 1, 1 ); 	
	dim3 dimBlock( blocksize, 1, 1 );
	GET_TIME(start);
    wordSearchSimple<<<dimGrid, dimBlock>>>(CharacterArray, numbytes, SuckMyAss, 3, IntArray);
    hipMemcpy( count, IntArray, intsize, hipMemcpyDeviceToHost );
	GET_TIME(finish);
	
	elapsed = finish - start;
    hipFree( CharacterArray );
	free(BookData);
	printf( "time is : %f \n number of occurances : %d " , elapsed, count[0] ); 
	 
 	return EXIT_SUCCESS;
}
