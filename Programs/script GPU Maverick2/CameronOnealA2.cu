//*****************************************************************
// Assignment #2
// Name: Alex Trampert and Cameron O'Neal
// GPU Programming Date: Date of Submission (10/16/2020)
//*****************************************************************
// The purpose of this progam is to solve for complex numbers
// using the Fast Fourier Transform(FFT). The FFT paired with blocks
// and thread partitions allows for the gpu to divide the data to increase 
// processing speed. We input a set of 8 complex and Imaginary numbers 
// filling the rest of our array values with 0. The program is to take 
// complex numbers, partition specific threads to solve the FFT odds and evens.
// The solving of the FFT odds and evens provides us with a Fourier Coeificient
// as the final result.
//*****************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>

//initializing basics values
const int N = 8192;
const int blocksize = 1024;

//******************************************************************
// function Name::computeEven()
// Parameters: 
// *xr  - initial reals array pointer 
// *xi  - initial imaginaries array pointer
// *fr  - final reals array pointer 
// *fi  - final imaginaries array pointer
// computes the even portion of the FFT
// using the various threads and dimensions
//********************************************************************
__global__
void computeEven(double *xr, double *xi, double *fr, double *fi)
{	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	/*
	//loop 4 times using j
	double _Complex z1 = xr[j] + xi[j] * I;
	double temp1 = cos(2 * 3.1415 * (j/8));
	double temp2 = (sin(2 * 3.1415 * (j/8)) * -1);
	double _Complex z2 = temp1 + temp2 * I;
	double _Complex product = z1 * z2;
	*/
	//Trying to use complex library so the computer can keep track
	//of real and imaginary but it will not recognize any form of
	//I we use (_I, _Complex_I, i)
	//j is the iteration we are on (evens - 0-2-4-6)
	//would store calculated real portion in fr[j] and imag portion in fi[j]
}
//******************************************************************
// function Name::computeOdd()
// Parameters: 
// *xr  - initial reals array pointer 
// *xi  - initial imaginaries array pointer
// *fr  - final reals array pointer 
// *fi  - final imaginaries array pointer
// computes the odd portion of the FFT
// using the various threads and dimensions
//********************************************************************
__global__
void computeOdd(double *xr, double *xi, double *fr, double *fi)
{	int i = ((blockIdx.x * blockDim.x + threadIdx.x) * 2) + 1;
	/*
	//loop 4 times using j
	double _Complex z1 = xr[j] + xi[j] * I;
	double temp1 = cos(2 * 3.1415 * (j/8));
	double temp2 = (sin(2 * 3.1415 * (j/8)) * -1);
	double _Complex z2 = temp1 + temp2 * I;
	double _Complex product = z1 * z2;
	*/
	
	//Trying to use complex library so the computer can keep track
	//of real and imaginary but it will not recognize any form of
	//I we use (_I, _Complex_I, i)
	//j is the iteration we are on (odds - 1-3-5-7)
	//would store calculated real portion in fr[j] and imag portion in fi[j]
}

int main()
{	//initializing arrays and sum integer
	double xr[N];
	double xi[N];
	double fr[N];
	double fi[N];
	double totalr = 0, totali = 0;

	//pointers for passing arrays
	double *xrd;
	double *xid;
	double *frd;
	double *fid;
	
	//hard coding the first 8 samples
	xr[0] = 3.6; xi[0] = 2.6;
	xr[1] = 2.9; xi[1] = 6.3;
	xr[2] = 5.6; xi[2] = 4.0;
	xr[3] = 4.8; xi[3] = 9.1;
	xr[4] = 3.3; xi[4] = 0.4;
	xr[5] = 5.9; xi[5] = 4.8;
	xr[6] = 5.0; xi[6] = 2.6;
	xr[7] = 4.3; xi[7] = 4.1;

	//filling arrays with the rest of samples
	for(int i = 8; i < N; i++)
    	{  xr[i] = 0;
		   xi[i] = 0;
    	}
	for(int i = 0; i < N; i++)
		{  fr[i] = 0;
		   fi[i] = 0;
		}

 	//value for data size
 	const double isize = N*sizeof(double);
	
	//allocating pointers with isize
 	hipMalloc( (void**)&xrd, isize );
	hipMalloc( (void**)&xid, isize );
	hipMalloc( (void**)&frd, isize );
	hipMalloc( (void**)&fid, isize );
 	
	//data transfers to function
 	hipMemcpy( xrd, xr, isize, hipMemcpyHostToDevice );
	hipMemcpy( xid, xi, isize, hipMemcpyHostToDevice );
	hipMemcpy( frd, fr, isize, hipMemcpyHostToDevice );
	hipMemcpy( fid, fi, isize, hipMemcpyHostToDevice );

	//4 blocks each consisting of 1024 threads
 	dim3 dimGrid( 4, 1 ); 	
	dim3 dimBlock( blocksize, 1 );

	//calling function with 4096 threads to work on evens
 	computeEven<<<dimGrid, dimBlock>>>(xrd, xid, frd, fid);

	//calling function with 4096 threads to work on odds
 	computeOdd<<<dimGrid, dimBlock>>>(xrd, xid, frd, fid);

	//data transfer to get new information in arrays from functions
 	hipMemcpy( fr, frd, isize, hipMemcpyDeviceToHost );
	hipMemcpy( fi, fid, isize, hipMemcpyDeviceToHost );
 	hipFree( xrd );
	hipFree( xid );
	hipFree( frd );
	hipFree( fid );

	//formatting output and summing totals of samples
	printf("TOTAL PROCESSED SAMPLES: %d\n", N);
	for(int i = 0; i < N/8; i++)
    	{  	totalr = 0; totali = 0;
			for(int j = 0; j < 8; j++)
			{ totalr += fr[j * i];
			  totali += fi[j * i];
			}
			printf("========================================\n");
			printf("XR[%d]: %f   XI[%d]: %f\n", i, totalr, i, totali);
    	}
	//sums up total for calculated real and imaginary arrays
	//groups them into samples of 8 numbers as in Table 1.Data of Time-Domain
	
 	return EXIT_SUCCESS;
}